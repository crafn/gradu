
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h> /* memcpy */
#include <math.h>
#include <stdint.h>

void *cuda_upload_var(void *host_var, int size)
{
	void *cuda_var;
	hipMalloc(&cuda_var, 4);
	hipMemcpy(cuda_var, host_var, size, hipMemcpyHostToDevice);
	return cuda_var;
}
void cuda_download_var(void *cuda_var, void *host_var, int size)
{
	hipMemcpy(host_var, cuda_var, size, hipMemcpyDeviceToHost);
	hipFree(cuda_var);
}

typedef struct intfield1
{
    int *m;
    int size[1];
    int is_device_field;
} intfield1;

intfield1 alloc_field_intfield1(int size_0)
{
    intfield1 field;
    field.m = (int*)malloc((sizeof(*field.m))*size_0);
    field.size[0] = size_0;
    field.is_device_field = 0;
    return field;
}

intfield1 alloc_device_field_intfield1(int size_0)
{
    intfield1 field;
    hipMalloc((void**)&field.m, (sizeof(*field.m))*size_0);
    field.size[0] = size_0;
    field.is_device_field = 1;
    return field;
}

void free_field_intfield1(intfield1 field)
{
    free(field.m);
}

void free_device_field_intfield1(intfield1 field)
{
    hipFree(field.m);
}

void memcpy_field_intfield1(intfield1 dst, intfield1 src)
{
    if (dst.is_device_field == 0 && src.is_device_field == 0) {
        hipMemcpy(dst.m, src.m, (sizeof(*dst.m))*dst.size[0], hipMemcpyHostToHost);
    }
    if (dst.is_device_field == 1 && src.is_device_field == 0) {
        hipMemcpy(dst.m, src.m, (sizeof(*dst.m))*dst.size[0], hipMemcpyHostToDevice);
    }
    if (dst.is_device_field == 0 && src.is_device_field == 1) {
        hipMemcpy(dst.m, src.m, (sizeof(*dst.m))*dst.size[0], hipMemcpyDeviceToHost);
    }
    if (dst.is_device_field == 1 && src.is_device_field == 1) {
        hipMemcpy(dst.m, src.m, (sizeof(*dst.m))*dst.size[0], hipMemcpyDeviceToDevice);
    }
}

int size_intfield1(intfield1 field, int index)
{
    return field.size[index];
}

typedef struct intmat1
{
    int m[1];
} intmat1;

int printf(const char *fmt, ...); /* TODO: Remove */

typedef intfield1 Field; /* One-dimensional integer field type */
__global__ void kernel_0(intfield1 *cuda_a, intfield1 b)
{
    intmat1 id;
    id.m[1*0] = (threadIdx.x + blockIdx.x*blockDim.x) % (*cuda_a).size[0]/1;
    (*cuda_a).m[1*id.m[1*0]] += b.m[1*id.m[1*0]];
}


int main()
{

    int N = 5;

    /* TODO: alloc_field -> alloc_host_field */

    Field a_data = alloc_field_intfield1(N);

    Field b_data = alloc_field_intfield1(N);
    a_data.m[1*0] = 1;
    a_data.m[1*1] = 2;
    a_data.m[1*2] = 3;
    a_data.m[1*3] = 4;
    a_data.m[1*4] = 5;
    b_data.m[1*0] = 10;
    b_data.m[1*1] = 20;
    b_data.m[1*2] = 30;
    b_data.m[1*3] = 40;
    b_data.m[1*4] = 50;

    Field a = alloc_device_field_intfield1(N);

    Field b = alloc_device_field_intfield1(N);
    memcpy_field_intfield1(a, a_data);
    memcpy_field_intfield1(b, b_data);

    {
        intfield1 *cuda_a = (intfield1*)cuda_upload_var(&a, sizeof(a));
        dim3 dim_grid(100, 1, 1);
        dim3 dim_block(a.size[0]/100, 1, 1);
        kernel_0<<<dim_grid, dim_block>>>(cuda_a, b);
        cuda_download_var(cuda_a, &a, sizeof(a));
    }
    memcpy_field_intfield1(a_data, a);

    for (int i = 0; i < N; ++i) {
        printf("%i ", a_data.m[1*i]);
    }
    free_field_intfield1(a_data);
    free_field_intfield1(b_data);
    free_device_field_intfield1(a);
    free_device_field_intfield1(b);

    return 0;
}

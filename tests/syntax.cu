
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h> /* memcpy */

/* Extended C syntax test
 */

/**********************/
/* BLOCK COMMENT TEST */
/**********************/
int aaaaa; /* Comment explaining 'aaaaa' */ /* See how comments are preserved in the output
 */
char bbbbbb;

typedef struct Aggregate2
{
    int c;
} Aggregate2;

typedef struct Aggregate
{
    int a;
    int b;
    Aggregate2 c;
} Aggregate;

/* These will be in global scope in the output code
 */
typedef struct LocalType
{
    int foo;
} LocalType;
void local_func(int p)
{
    LocalType bug_test; /* Just to test that identifier lookup works
 */
    bug_test.foo = 123;
    aaaaa = bug_test.foo + p;
}
int main(int argc, const char **argv)
{
    int temp_var;
    int i;

    Aggregate foo1 = {
        0,
        1,
        { 2 }
    };
    temp_var = 1 + 2 * 3;
    local_func(10);

    {
        int test;
        test = 5;
        aaaaa = test;
    }

    if (1) {
        /* Foo */
        for (i = 0; i < 10; i = i + 1) {
            temp_var = temp_var + 1;
        }
    } else if (2) {
        for (i = 0; i < 10; i = i + 1) 
            ;
    } else {
        /* Bar */
        if (1) {
            i = 2;
        }
        while (i) {
            i = i - 1;
        }
    }

    return foo1.a;
}


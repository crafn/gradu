
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h> /* memcpy */
#include <math.h>
#include <stdint.h>

void *cuda_upload_var(void *host_var, int size)
{
	void *cuda_var;
	hipMalloc(&cuda_var, 4);
	hipMemcpy(cuda_var, host_var, size, hipMemcpyHostToDevice);
	return cuda_var;
}
void cuda_download_var(void *cuda_var, void *host_var, int size)
{
	hipMemcpy(host_var, cuda_var, size, hipMemcpyDeviceToHost);
	hipFree(cuda_var);
}

int printf(const char *fmt, ...);

int main(int argc, char **argv)
{
    printf("Hello World!\n");

    return 0;
}
